#include <hip/hip_runtime.h>

#include <iostream>

int main() {
    int num_devs, driver_version;
    hipGetDeviceCount(&num_devs);
    std::cout << "This machine has " << num_devs << " devices." << std::endl;

    hipDriverGetVersion(&driver_version);
    std::cout << "The version of cuda of this machine is " << driver_version << std::endl;
    hipDevice_t cu_dev;
    hipCtxGetDevice(&cu_dev);
    int attr;
    hipDeviceGetAttribute(&attr, CU_DEVICE_ATTRIBUTE_GPU_DIRECT_RDMA_FLUSH_WRITES_OPTIONS, cu_dev);
    if (attr & hipFlushGPUDirectRDMAWritesOptionHost) {
        std::cout << "This machine support GDR write." << std::endl;
    }
    else {
        std::cout << "This machine does not support GDR write." << std::endl;
    }
}