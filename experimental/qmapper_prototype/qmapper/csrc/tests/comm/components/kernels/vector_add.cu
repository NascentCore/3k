#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include "kernels/vector_add.h"

template<typename T>
__global__ void matAdd_cuda(T *a,T *b,T *sum)
{
    int i = blockIdx.x*blockDim.x+ threadIdx.x;
    sum[i] = a[i] + b[i];
}


hipError_t matAdd(float *a,float *b,float *sum, int length)
{
    int device = 0;
    hipSetDevice(device);
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, device);
    int threadMaxSize = devProp.maxThreadsPerBlock;
    int blockSize = (length+threadMaxSize-1)/threadMaxSize;
    dim3 block(blockSize);
    int size = length * sizeof(float);
    float *sumGPU,*aGPU,*bGPU;
    hipMalloc((void**)&sumGPU,size);
    hipMalloc((void**)&aGPU,size);
    hipMalloc((void**)&bGPU,size);
    hipMemcpy((void*)aGPU,(void*)a,size,hipMemcpyHostToDevice);
    hipMemcpy((void*)bGPU,(void*)b,size,hipMemcpyHostToDevice);
    matAdd_cuda<float><<<block,size/blockSize>>>(aGPU,bGPU,sumGPU);
    hipMemcpy(sum,sumGPU,size,hipMemcpyDeviceToHost);
    hipFree(sumGPU);
    hipFree(aGPU);
    return hipFree(bGPU);
}