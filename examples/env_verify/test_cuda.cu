
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int a, int b, int *c) {
    *c = a + b;
}

int main() {
    int c;
    int *dev_c;
    hipMalloc((void **)&dev_c, sizeof(int));
    add<<<1,1>>>(2, 7, dev_c);
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_c);
    printf("2 + 7 = %d\n", c);
    return 0;
}
